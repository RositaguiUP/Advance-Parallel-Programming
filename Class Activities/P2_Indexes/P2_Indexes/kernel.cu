#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

using namespace std;

__global__ void idx_calc_tid(int* input)
{
    int tid = threadIdx.x;
    printf("[DEVICE] threadIdx.x: %d, data: %d\n\r", tid, input[tid]);
}

__global__ void idx_calc_gid(int* input)
{
    int tid = threadIdx.x;
    int block_offset = blockDim.x * blockIdx.x;
    int gid = tid + block_offset;
    printf("[DEVICE] blockIdx.x: %d, threadIdx.x: %d, gId: %d, data: %d\n\r", blockIdx.x, tid, gid, input[gid]);
}

__global__ void idx_calc_2d(int* input)
{
    int tid = threadIdx.x;
    int block_offset = blockDim.x * blockIdx.x;

    int threads_per_row = gridDim.x * blockDim.x;
    int row_offset = blockIdx.y * threads_per_row;
    int gid = tid + row_offset + block_offset;
    printf("[DEVICE] gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, gId: %d, data: %d\n\r",
        gridDim.x, blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

__global__ void idx_calc_2d_2d(int* input)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y;
    int block_offset = blockIdx.x * threads_per_block;

    int threads_per_row = gridDim.x * threads_per_block;
    int row_offset = blockIdx.y * threads_per_row;

    int gid = tid + row_offset + block_offset;
    printf("[DEVICE] gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d, threadidx.x: %d, gId: %d, data: %d\n\r",
        gridDim.x, blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}


__global__ void idx_calc_3d(int* input)
{
    int tid = blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z) + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y * blockDim.z;
    int block_offset = blockIdx.x * threads_per_block;

    int threads_per_row = gridDim.x * threads_per_block;
    int threads_per_face = gridDim.y * threads_per_row;
    int row_offset = blockIdx.y * threads_per_row;
    int face_offset = blockIdx.z * threads_per_face;
    
    int gid = tid + row_offset + block_offset + face_offset;

    // int bid = gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z) + blockIdx.x;
    // int gid = tid + bid * threads_per_block;
    printf("[DEVICE] gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadidx.x: %d, gId: %d, data: %d\n\r",
        gridDim.x, blockIdx.x, blockIdx.y, blockIdx.z, tid, gid, input[gid]);
}


int main()
{
    const int vectorSize = 64; // 16 16 16 32
    int* dev_a;

    hipMalloc((void**)&dev_a, vectorSize * sizeof(int));

    int* phost_a;

    phost_a = (int*)malloc(vectorSize * sizeof(int));

    for (int i = 0; i < vectorSize; i++) {
        phost_a[i] = i;
        printf("[HOST] data: %d\n\r", phost_a[i]);
    }

    hipMemcpy(dev_a, phost_a, vectorSize * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(2, 2, 2); //  (16) (8) (4) (4,2);
    dim3 gridDim(2, 2, 2);  //  (1) (2) (2,2) (2,2);

    // idx_calc_tid << < gridDim, blockDim >> > (dev_a);
    // idx_calc_gid << < gridDim, blockDim >> > (dev_a);
    // idx_calc_2d << < gridDim, blockDim >> > (dev_a);
    // idx_calc_2d_2d << < gridDim, blockDim >> > (dev_a);
    idx_calc_3d << < gridDim, blockDim >> > (dev_a);

    hipDeviceSynchronize();

    hipDeviceReset();
    hipFree(dev_a);

    return 0;
}