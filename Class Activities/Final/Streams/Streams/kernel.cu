#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void simple_kernel() {
    printf("Hello from kernel");
}

__global__ void stream_test(int* in, int* out, int size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size) {
        // ANY CALC
        for (int i = 0; i < 25; i++) {
            out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
        }
    }
}


int main()
{
    int size = 1 << 18;
    int byte_size = size * sizeof(int);

    // Initialize host pointer
    int* h_in, * h_ref, * h_in2, * h_ref2;

    hipHostMalloc((void**)&h_in, byte_size);
    hipHostMalloc((void**)&h_ref, byte_size);
    hipHostMalloc((void**)&h_in2, byte_size);
    hipHostMalloc((void**)&h_ref2, byte_size);

    srand((double)time(NULL));
    for (int i = 0; i < size; i++) {
        h_in[i] = rand();
        h_in2[i] = rand();
    }

    // allocate device pointers
    int* d_in, * d_out, * d_in2, * d_out2;
    hipMalloc((void**)&d_in, byte_size);
    hipMalloc((void**)&d_out, byte_size);
    hipMalloc((void**)&d_in2, byte_size);
    hipMalloc((void**)&d_out2, byte_size);

    // kernel launch
    dim3 block(128);
    dim3 grid(size / block.x);
    hipStream_t str, str2;
    hipStreamCreate(&str);
    hipStreamCreate(&str2);

    // trasfer data from host to device
    hipMemcpyAsync(d_in, h_in, byte_size, hipMemcpyHostToDevice, str);
    stream_test <<< grid, block, 0, str >> > (d_in, d_out, size);
    hipMemcpyAsync(h_ref, d_out, byte_size, hipMemcpyDeviceToHost, str);

    hipMemcpyAsync(d_in2, h_in2, byte_size, hipMemcpyHostToDevice, str2);
    stream_test <<< grid, block, 0, str2 >> > (d_in, d_out, size);
    hipMemcpyAsync(h_ref2, d_out2, byte_size, hipMemcpyDeviceToHost, str2);

    hipStreamSynchronize(str);
    hipStreamDestroy(str);

    hipStreamSynchronize(str2);
    hipStreamDestroy(str2);

    hipDeviceReset();
    /*int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    if (&deviceProp.concurrentKernels == 0) {
        printf("> GPU doesn not support concurrent kernel execution \n");
        printf("kernel executiion will be serialized \n");
    }

    hipStream_t str1, str2, str3;

    hipStreamCreate(&str1);
    hipStreamCreate(&str2);
    hipStreamCreate(&str3);

    simple_kernel <<< 1, 1, 0, str1 >> > ();
    simple_kernel <<< 1, 1, 0, str2 >> > ();
    simple_kernel <<< 1, 1, 0, str3 >> > ();

    hipStreamDestroy(str1);
    hipStreamDestroy(str2);
    hipStreamDestroy(str3);

    hipDeviceSynchronize();
    hipDeviceReset();*/

    return 0;
}

/*
P1: SUma de arreglos
N = 8 streams

*/
