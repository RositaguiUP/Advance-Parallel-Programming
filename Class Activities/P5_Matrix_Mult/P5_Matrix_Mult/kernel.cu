#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <ctime>

using namespace std;

__global__ void dotProduct(int* a, int* b, int* c, int dimN)
{
    int tid = blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z) + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y * blockDim.z;
    int bid = gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z) + blockIdx.x;
    int gid = tid + bid * threads_per_block;
    
    c[gid] = 0;

    int aStart = (int)(gid / dimN)* dimN;
    int bStart = (int)(gid % dimN);

    for (int i = 0; i < dimN; i++) {
        int col = aStart + i;
        int row = bStart + i * dimN;
        c[gid] += a[col] * b[row];
    }
}

void printMatrix(int* a, int dimN) {
    for (int i = 0; i < dimN * dimN; i++) {
        if (i % dimN == 0) {
            printf("\n");
        }
        printf("\t%d", a[i]);
    }
}

int main()
{
    const int vectorSize = 4;
    const int size = vectorSize * sizeof(int);
    int dimN = 2;
    int* dev_a, * dev_b, * dev_c;

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    int* phost_a, * phost_b, * phost_c;

    phost_a = (int*)malloc(size);
    phost_b = (int*)malloc(size);
    phost_c = (int*)malloc(size);

    for (int i = 0; i < vectorSize; i++) {
        phost_a[i] = i + 1;
        phost_b[i] = i + 1 + vectorSize;
    }

    hipMemcpy(dev_a, phost_a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, phost_b, size, hipMemcpyHostToDevice);

    dim3 blockDim(2, 2);
    dim3 gridDim(1);

    clock_t gpu_start, gpu_stop;

    gpu_start = clock();
    dotProduct << < gridDim, blockDim >> > (dev_a, dev_b, dev_c, dimN);
    hipDeviceSynchronize();

    gpu_stop = clock();
    double cps_gpu = (double)((double)(gpu_stop - gpu_start) / CLOCKS_PER_SEC);
    printf("\n\nExecution Time [ET.GPU]: %4.6f\n\r", cps_gpu);

    hipMemcpy(phost_c, dev_c, size, hipMemcpyDeviceToHost);

    printf("\n\n*****    MATRIX A    *****\n");
    printMatrix(phost_a, dimN);

    printf("\n\n*****    MATRIX B    *****\n");
    printMatrix(phost_b, dimN);

    printf("\n\n*****    MATRIX C    *****\n");
    printMatrix(phost_c, dimN);

    hipDeviceReset();
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
